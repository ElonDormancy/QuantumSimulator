﻿#include "hip/hip_runtime.h" 
#include "hipblas.h" 
#include <iostream>
#include "datatypes.h"
#include "MulcuBLAS.cuh"
#include <complex>
using namespace std;

extern "C" MatrixC MatrixMul(gate Gate, MatrixC mat, u_int N)
{
    // 定义状态变量 
    hipblasStatus_t status;
    // 在内存中为将要计算的矩阵开辟空间 
    hipDoubleComplex* h_A = (hipDoubleComplex*)malloc(2 * 2 * sizeof(hipDoubleComplex));
    hipDoubleComplex* h_B = (hipDoubleComplex*)malloc(2 * N * sizeof(hipDoubleComplex));
    // 在内存中为将要存放运算结果的矩阵开辟空间 
    hipDoubleComplex* h_C = (hipDoubleComplex*)malloc(2 * N * sizeof(hipDoubleComplex));
    // 打印待测试的矩阵 
    int k_gate = 0;
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            h_A[k_gate] = make_hipDoubleComplex(real(Gate(i,j)), imag(Gate(i,j)));
            k_gate++;
        }
    }
    int k_mat = 0;
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_B[k_mat] = make_hipDoubleComplex(real(mat(i, j)), imag(mat(i, j)));
            k_mat++;
        }
    }
    /*
        ** GPU 计算矩阵相乘
        */
        // 创建并初始化 CUBLAS 库对象
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
            cout << "CUBLAS 对象实例化出错" << endl;
        }
        getchar();
    }
    hipDoubleComplex* d_A, * d_B, * d_C;
    // 在 显存 中为将要计算的矩阵开辟空间 
    hipMalloc(
        (void**)&d_A, // 指向开辟的空间的指针 
        2 * 2 * sizeof(hipDoubleComplex) //　需要开辟空间的字节数 
    );
    hipMalloc(
        (void**)&d_B,
        2 * N * sizeof(hipDoubleComplex)
    );
    // 在 显存 中为将要存放运算结果的矩阵开辟空间 
    hipMalloc(
        (void**)&d_C,
        2 * N * sizeof(hipDoubleComplex)
    );
    // 将矩阵数据传递进 显存 中已经开辟好了的空间 
    hipblasSetVector(
        2 * 2, // 要存入显存的元素个数 
        sizeof(hipDoubleComplex), // 每个元素大小 
        h_A, // 主机端起始地址 
        1, // 连续元素之间的存储间隔 
        d_A, // GPU 端起始地址 
        1 // 连续元素之间的存储间隔 
    );
    hipblasSetVector(
        2 * N,
        sizeof(hipDoubleComplex),
        h_B,
        1,
        d_B,
        1
    );
    // 同步函数
    hipDeviceSynchronize();
    // 传递进矩阵相乘函数中的参数，具体含义请参考函数手册。 
    hipDoubleComplex a = make_hipDoubleComplex(1.0, 0); hipDoubleComplex b = make_hipDoubleComplex(0, 0);
    // 矩阵相乘。该函数必然将数组解析成列优先数组 
    hipblasZgemm(
        handle, // blas 库对象 
        HIPBLAS_OP_N, // 矩阵 A 属性参数 
        HIPBLAS_OP_N, // 矩阵 B 属性参数
        N, // A, C 的行数 
        2, // B, C 的列数
        2, // A 的列数和 B 的行数
        &a, // 运算式的 α 值 
        d_B, // A 在显存中的地址 
        N, // lda 
        d_A, // B 在显存中的地址 
        2, // ldb 
        &b, // 运算式的 β 值 
        d_C, // C 在显存中的地址(结果矩阵) 
        N // 
    );
    // 同步函数 
    hipDeviceSynchronize();
    // 从 显存 中取出运算结果至 内存中去
    hipblasGetVector(2 * N, // 要取出元素的个数 
        sizeof(hipDoubleComplex), // 每个元素大小 
        d_C, // GPU 端起始地址 
        1, // 连续元素之间的存储间隔 
        h_C, // 主机端起始地址
        1 // 连续元素之间的存储间隔 
    );
    MatrixC mat_o = MatrixC::Zero(2, N);
    int k_o = 0;
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < N; j++)
        {
            mat_o(i,j) = { hipCreal(h_C[k_o]) ,hipCimag(h_C[k_o]) };
        }
    };
    // 清理掉使用过的内存 
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // 释放 CUBLAS 库对象
    hipblasDestroy(handle);
    return mat_o;
}